#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdexcept>
#include <hip/hip_runtime.h>
#include <npp.h>
#include <hip/hip_runtime_api.h>

#include <ImagesCPU.h>
#include <ImagesNPP.h>

#include <mex.h>

inline int cudaDeviceInit()
{
    int deviceCount;
    checkCudaErrors(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0)
    {
        fprintf(stderr, "CUDA error: no devices supporting CUDA.\n");
        exit(EXIT_FAILURE);
    }

    int dev = 0;

    if (dev > deviceCount-1)
    {
        fprintf(stderr, ">> %d CUDA capable GPU device(s) detected. <<\n", deviceCount);
        fprintf(stderr, ">> cudaDeviceInit (-device= %d) is not a valid GPU device. <<\n\n", dev);
        return -dev;
    }
    else
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);
        fprintf(stderr, "hipSetDevice GPU %d = %s\n", dev, deviceProp.name);
    }

    return dev;
}

int printfNPPinfo(int cudaVerMajor, int cudaVerMinor)
{
    fprintf(stderr, "Getting npp version\n");
    const NppLibraryVersion *libVer   = nppGetLibVersion();

    fprintf(stderr, "NPP Library Version %d.%d.%d\n", libVer->major, libVer->minor, libVer->build);

    int driverVersion, runtimeVersion;
    hipDriverGetVersion(&driverVersion);
    hipRuntimeGetVersion(&runtimeVersion);

    fprintf(stderr, "  CUDA Driver  Version: %d.%d\n", driverVersion/1000, (driverVersion%100)/10);
    fprintf(stderr, "  CUDA Runtime Version: %d.%d\n", runtimeVersion/1000, (runtimeVersion%100)/10);

    int bVal = checkCudaCapabilities(cudaVerMajor, cudaVerMinor);
    return bVal;
}

__global__ void updateSourceCapacity(Npp32s* terminals, int pitch, Npp32s lambda, int nSourceVertices, int* irow, int* icol)
{
    int i = blockIdx.x;
    if (i < nSourceVertices)
    {
        Npp32s *p = (Npp32s*)((char*)terminals + irow[i]*pitch) + icol[i];
        *p -= lambda;
    }
}

__global__ void updateSinkCapacity(Npp32s* terminals, int pitch, Npp32s lambda, int nSinkVertices, int* irow, int* icol)
{
    int i = blockIdx.x;
    if (i < nSinkVertices)
    {
        Npp32s *p = (Npp32s*)((char*)terminals + irow[i]*pitch) + icol[i];
        *p += lambda;  
    }
}

__global__ void updateCapacity(Npp32s* terminals, int pitch, Npp32s lambda, int width, int height)
{
    int row = blockIdx.x;
    int col = blockIdx.y;
    if (row < height && col < width)
    {
        Npp32s *p = (Npp32s*)((char*)terminals + row*pitch) + col;
        *p += lambda;
    }
}

void graphCut(int width, int height, Npp32s *pTerminals, Npp32s *pLeftTransposed,
    Npp32s *pRightTransposed, Npp32s *pTop, Npp32s *pBottom, Npp8u *labels, int &distinctCuts,
    int nLambdas, Npp32s* lambdas, Npp32s* distinctLambdas, int nSourceVertices, int* sourceVertices, int nSinkVertices, int* sinkVertices)
{
    // fprintf(stderr, "Starting cuda graphcut computation on image of size %d x %d...\n\n", width, height);

    // cudaDeviceInit();

    // // Min spec is SM 1.1 devices
    // if (!printfNPPinfo(1, 1))
    // {
    //     fprintf(stderr, "Insufficient Compute Capability (must be >= 1.1)\n");
    //     hipDeviceReset();
    //     exit(EXIT_SUCCESS);
    // }


    NppiSize size;
    size.width = width;
    size.height = height;

    //Alocate memory on the device
    Npp32s *d_terminals;
    Npp32s *d_left_transposed, *d_right_transposed;
    Npp32s *d_top, *d_bottom;
    size_t step, transposed_step;

    hipEvent_t copy_start, copy_stop;
    hipEventCreate(&copy_start);
    hipEventCreate(&copy_stop);

    // Compute the graphcut, result is 0 / !=0
    hipEventRecord(copy_start,0);
    
    checkCudaErrors(hipMallocPitch(&d_terminals, &step, width*sizeof(Npp32s), height));
    checkCudaErrors(hipMallocPitch(&d_top, &step, width*sizeof(Npp32s), height));
    checkCudaErrors(hipMallocPitch(&d_bottom, &step, width*sizeof(Npp32s), height));
    checkCudaErrors(hipMallocPitch(&d_left_transposed, &transposed_step, height*sizeof(Npp32s), width));
    checkCudaErrors(hipMallocPitch(&d_right_transposed, &transposed_step, height*sizeof(Npp32s), width));

    //Copy capacities to device
    checkCudaErrors(hipMemcpy2D(d_terminals, step, pTerminals, width * sizeof(Npp32s), width*sizeof(Npp32s), height, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy2D(d_top,       step, pTop,       width * sizeof(Npp32s), width*sizeof(Npp32s), height, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy2D(d_bottom,    step, pBottom,    width * sizeof(Npp32s), width*sizeof(Npp32s), height, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy2D(d_left_transposed,  transposed_step, pLeftTransposed, height * sizeof(Npp32s), height*sizeof(Npp32s), width, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy2D(d_right_transposed, transposed_step, pRightTransposed, height * sizeof(Npp32s), height*sizeof(Npp32s), width, hipMemcpyHostToDevice));

    // Allocate temp storage for graphcut computation
    Npp8u *pBuffer;
    int bufferSize;
    nppiGraphcutGetSize(size, &bufferSize);
    checkCudaErrors(hipMalloc(&pBuffer, bufferSize));

    NppiGraphcutState *pGraphcutState;
    nppiGraphcutInitAlloc(size, &pGraphcutState, pBuffer);

     // Allocate label storage
    npp::ImageNPP_8u_C1 oDeviceDst(width, height);

    // declare a host image object for an 8-bit grayscale image
    npp::ImageCPU_8u_C1 oHostAlpha(width, height);

    npp::ImageNPP_8u_C1 oDeviceAlpha(width, height);

    int* sourceRows = new int[nSourceVertices];
    int* sourceCols = new int[nSourceVertices];
    for (int i = 0; i < nSourceVertices; ++i)
    {
        sourceRows[i] = (sourceVertices[i]-1)%height;
        sourceCols[i] = (sourceVertices[i]-1)/height;
    }

    int *d_sourceRows, *d_sourceCols;
    checkCudaErrors(hipMalloc(&d_sourceRows, nSourceVertices*sizeof(int)));
    checkCudaErrors(hipMemcpy(d_sourceRows, sourceRows, nSourceVertices*sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc(&d_sourceCols, nSourceVertices*sizeof(int)));
    checkCudaErrors(hipMemcpy(d_sourceCols, sourceCols, nSourceVertices*sizeof(int), hipMemcpyHostToDevice));

    int* sinkRows = new int[nSinkVertices];
    int* sinkCols = new int[nSinkVertices];
    for (int i = 0; i < nSinkVertices; ++i)
    {
        sinkRows[i] = (sinkVertices[i]-1)%height;
        sinkCols[i] = (sinkVertices[i]-1)/height;
    }

    int *d_sinkRows, *d_sinkCols;
    checkCudaErrors(hipMalloc(&d_sinkRows, nSinkVertices*sizeof(int)));
    checkCudaErrors(hipMemcpy(d_sinkRows, sinkRows, nSinkVertices*sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc(&d_sinkCols, nSinkVertices*sizeof(int)));
    checkCudaErrors(hipMemcpy(d_sinkCols, sinkCols, nSinkVertices*sizeof(int), hipMemcpyHostToDevice));

    hipEventRecord(copy_stop,0);
    hipEventSynchronize(copy_stop);

    float copy_time;
    hipEventElapsedTime(&copy_time, copy_start, copy_stop);
    fprintf(stderr, "Copy elapsed Time:  %f ms\n", copy_time);


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Compute the graphcut, result is 0 / !=0
    hipEventRecord(start,0);

    int iout = 0;
    distinctCuts = 0;
    for (int k = 0; k < nLambdas; ++k)
    {
        Npp32s lambda = (k > 0) ? (lambdas[k]-lambdas[k-1]) : lambdas[0];

        //update d_terminals
        updateSourceCapacity<<<nSourceVertices, 1>>>(d_terminals, step, lambda, nSourceVertices, d_sourceRows, d_sourceCols);
        updateSinkCapacity<<<nSinkVertices, 1>>>(d_terminals, step, lambda, nSinkVertices, d_sinkRows, d_sinkCols);
        // updateCapacity<<<height, width>>>(d_terminals, step, lambda, width, height);

        NPP_CHECK_NPP(nppiGraphcut_32s8u(d_terminals, d_left_transposed, d_right_transposed,
                       d_top, d_bottom, step, transposed_step,
                       size, oDeviceDst.data(), oDeviceDst.pitch(), pGraphcutState));
        // printf("%s\n", hipGetErrorString(hipGetLastError()) );
        // printf("graphcut done\n");

        // convert graphcut result to 0/255 alpha image using new nppiCompareC_8u_C1R primitive (CUDA 5.0)
        NPP_CHECK_NPP(nppiCompareC_8u_C1R(oDeviceDst.data(), oDeviceDst.pitch(), 0, oDeviceAlpha.data(), oDeviceAlpha.pitch(), size,
                            NPP_CMP_GREATER));

        // and copy the result to host
        oDeviceAlpha.copyTo(oHostAlpha.data(), oHostAlpha.pitch());

        bool is_distinct = true;
        if (iout > 0)
        {
            bool ok = true;
            for (int j = 0, icrn = 0; j < width && ok; ++j)
                for (int i = 0; i < height && ok; ++i)
                {
                    Npp8u val = *oHostAlpha.data(j, i) ? 1:0;
                    if (val != labels[iout-width*height+icrn])
                        ok = false;
                    icrn++;
                }
            is_distinct = !ok;
        }

        if (is_distinct)
        {
            for (int j = 0; j < width; ++j)
                for (int i = 0; i < height; ++i)
                    labels[iout++] = *oHostAlpha.data(j, i) ? 1:0;
            distinctLambdas[distinctCuts++] = lambdas[k];
        }
    }

    // printf("Distinct cuts: %d\n", distinctCuts);
    
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    float time;
    hipEventElapsedTime(&time, start, stop);
    fprintf(stderr, "Elapsed Time:  %f ms\n", time);

    delete [] sourceRows;
    delete [] sourceCols;
    delete [] sinkRows;
    delete [] sinkCols;
           
    checkCudaErrors(hipFree(d_terminals));
    checkCudaErrors(hipFree(d_top));
    checkCudaErrors(hipFree(d_bottom));
    checkCudaErrors(hipFree(d_left_transposed));
    checkCudaErrors(hipFree(d_right_transposed));
    checkCudaErrors(hipFree(pBuffer));
    checkCudaErrors(hipFree(d_sourceRows));
    checkCudaErrors(hipFree(d_sourceCols));
    checkCudaErrors(hipFree(d_sinkRows));
    checkCudaErrors(hipFree(d_sinkCols));
    nppiGraphcutFree(pGraphcutState);

    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
}

template<typename T>
T *transpose(T* mat, int rows, int cols)
{
    T* tmp = new T[rows*cols];
    for (int i = 0; i < rows; ++i)
        for (int j = 0; j < cols; ++j)
            tmp[i*cols+j] = mat[j*rows+i];
    return tmp;
}

template<typename T>
void transposeInPlace(T* mat, int rows, int cols)
{
    T* tmp = new T[rows*cols];
    int k = 0;
    for (int j = 0; j < cols; ++j)
        for (int i = 0; i < rows; ++i)
            tmp[k++] = mat[i*cols+j];
    memcpy(mat, tmp, sizeof(T)*rows*cols);
    delete [] tmp;
}

extern void mexFunction(int iNbOut, mxArray *pmxOut[],
    int iNbIn, const mxArray *pmxIn[])
{
    int width = mxGetN(pmxIn[0]);
    int height = mxGetM(pmxIn[0]);

    Npp32s *pTerminals = transpose((Npp32s*)mxGetData(pmxIn[0]), height, width);

    Npp32s* pLeftTransposed = transpose((Npp32s*)mxGetData(pmxIn[1]), width, height);
    for (int j = 0; j < height; ++j)
        if (pLeftTransposed[j] != 0)
            throw std::invalid_argument("pLeftTransposed[0][*] must be 0");

    Npp32s* pRightTransposed = transpose((Npp32s*)mxGetData(pmxIn[2]), width, height);
    for (int j = 0; j < height; ++j)
        if (pRightTransposed[(width-1)*height + j] != 0)
            throw std::invalid_argument("pRightTransposed[width-1][*] must be 0");  

    Npp32s* pTop = transpose((Npp32s*)mxGetData(pmxIn[3]), height, width);
    for (int j = 0 ; j < width; ++j)
        if (pTop[j] != 0)
            throw std::invalid_argument("pTop[0][*] must be 0");

    Npp32s* pBottom = transpose((Npp32s*)mxGetData(pmxIn[4]), height, width);
    for (int j = 0; j < width; ++j)
        if (pBottom[width*(height-1) + j] != 0)
            throw std::invalid_argument("pBottom[height-1][*] must be 0"); 

    fprintf(stderr, "Assertions passed\n");  

    int nLambdas = mxGetN(pmxIn[5]);
    Npp32s* lambdas = (Npp32s*)mxGetData(pmxIn[5]);

    int nSourceVertices = mxGetN(pmxIn[6]);
    int* sourceVertices = (int*)mxGetData(pmxIn[6]);

    int nSinkVertices = mxGetN(pmxIn[7]);
    int* sinkVertices = (int*)mxGetData(pmxIn[7]);

    Npp8u *outmat = new Npp8u[width*height*nLambdas];
    Npp32s *distinctLambdas = new Npp32s[nLambdas];

    int distinctCuts = 0;
    graphCut(width, height, pTerminals, pLeftTransposed, pRightTransposed, pTop, pBottom, outmat, distinctCuts,
        nLambdas, lambdas, distinctLambdas, nSourceVertices, sourceVertices, nSinkVertices, sinkVertices);

    pmxOut[0] = mxCreateNumericMatrix(height*width, distinctCuts, mxUINT8_CLASS, mxREAL);
    Npp8u *outp = (Npp8u*)mxGetPr(pmxOut[0]);
    memcpy(outp, outmat, height*width*distinctCuts*sizeof(Npp8u));

    pmxOut[1] = mxCreateNumericMatrix(1, distinctCuts,  mxINT32_CLASS, mxREAL);
    Npp32s *lambdap = (Npp32s*)mxGetPr(pmxOut[1]);
    memcpy(lambdap, distinctLambdas, distinctCuts*sizeof(Npp32s));

    // transposeInPlace(outmat, height, width); 

    delete [] pTerminals;
    delete [] pLeftTransposed;
    delete [] pRightTransposed;
    delete [] pTop;
    delete [] pBottom;
    delete [] outmat;
    delete [] distinctLambdas;
}
