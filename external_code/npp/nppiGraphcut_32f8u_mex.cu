#include <stdio.h>
#include <stdexcept>
#include <hip/hip_runtime.h>
#include <npp.h>
#include <hip/hip_runtime_api.h>

#include <ImagesCPU.h>
#include <ImagesNPP.h>

#include <mex.h>

inline int cudaDeviceInit()
{
    int deviceCount;
    checkCudaErrors(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0)
    {
        fprintf(stderr, "CUDA error: no devices supporting CUDA.\n");
        exit(EXIT_FAILURE);
    }

    int dev = 0;

    if (dev > deviceCount-1)
    {
        fprintf(stderr, ">> %d CUDA capable GPU device(s) detected. <<\n", deviceCount);
        fprintf(stderr, ">> cudaDeviceInit (-device= %d) is not a valid GPU device. <<\n\n", dev);
        return -dev;
    }
    else
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);
        fprintf(stderr, "hipSetDevice GPU %d = %s\n", dev, deviceProp.name);
    }

    return dev;
}

int printfNPPinfo(int cudaVerMajor, int cudaVerMinor)
{
    fprintf(stderr, "Getting npp version\n");
    const NppLibraryVersion *libVer   = nppGetLibVersion();

    fprintf(stderr, "NPP Library Version %d.%d.%d\n", libVer->major, libVer->minor, libVer->build);

    int driverVersion, runtimeVersion;
    hipDriverGetVersion(&driverVersion);
    hipRuntimeGetVersion(&runtimeVersion);

    fprintf(stderr, "  CUDA Driver  Version: %d.%d\n", driverVersion/1000, (driverVersion%100)/10);
    fprintf(stderr, "  CUDA Runtime Version: %d.%d\n", runtimeVersion/1000, (runtimeVersion%100)/10);

    int bVal = checkCudaCapabilities(cudaVerMajor, cudaVerMinor);
    return bVal;
}

void graphCut(int width, int height, Npp32f *pTerminals, Npp32f *pLeftTransposed,
Npp32f *pRightTransposed, Npp32f *pTop, Npp32f *pBottom, Npp8u *labels)
{
    fprintf(stderr, "Starting cuda graphcut computation on image of size %d x %d...\n\n", width, height);

    cudaDeviceInit();

    // Min spec is SM 1.1 devices
    if (!printfNPPinfo(1, 1))
    {
        fprintf(stderr, "Insufficient Compute Capability (must be >= 1.1)\n");
        hipDeviceReset();
        exit(EXIT_SUCCESS);
    }


    NppiSize size;
    size.width = width;
    size.height = height;

    //Alocate memory on the device
    Npp32f *d_terminals;
    Npp32f *d_left_transposed, *d_right_transposed;
    Npp32f *d_top, *d_bottom;
    size_t step, transposed_step;
    
    checkCudaErrors(hipMallocPitch(&d_terminals, &step, width*sizeof(Npp32f), height));
    checkCudaErrors(hipMallocPitch(&d_top, &step, width*sizeof(Npp32f), height));
    checkCudaErrors(hipMallocPitch(&d_bottom, &step, width*sizeof(Npp32f), height));
    checkCudaErrors(hipMallocPitch(&d_left_transposed, &transposed_step, height*sizeof(Npp32f), width));
    checkCudaErrors(hipMallocPitch(&d_right_transposed, &transposed_step, height*sizeof(Npp32f), width));

    fprintf(stderr, "Malloc done\n");


    //Copy capacities to device
    checkCudaErrors(hipMemcpy2D(d_terminals, step, pTerminals, width * sizeof(Npp32f), width*sizeof(Npp32f), height, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy2D(d_top,       step, pTop,       width * sizeof(Npp32f), width*sizeof(Npp32f), height, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy2D(d_bottom,    step, pBottom,    width * sizeof(Npp32f), width*sizeof(Npp32f), height, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy2D(d_left_transposed,  transposed_step, pLeftTransposed, height * sizeof(Npp32f), height*sizeof(Npp32f), width, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy2D(d_right_transposed, transposed_step, pRightTransposed, height * sizeof(Npp32f), height*sizeof(Npp32f), width, hipMemcpyHostToDevice));

    fprintf(stderr, "cudaMemcpy2d done\n");


    // Allocate temp storage for graphcut computation
    Npp8u *pBuffer;
    int bufferSize;
    nppiGraphcutGetSize(size, &bufferSize);
    checkCudaErrors(hipMalloc(&pBuffer, bufferSize));

    NppiGraphcutState *pGraphcutState;
    nppiGraphcutInitAlloc(size, &pGraphcutState, pBuffer);

     // Allocate label storage
    npp::ImageNPP_8u_C1 oDeviceDst(width, height);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Compute the graphcut, result is 0 / !=0
    hipEventRecord(start,0);

    fprintf(stderr, "Calling nppiGraphcut_32f8u\n");
    NPP_CHECK_NPP(nppiGraphcut_32f8u(d_terminals, d_left_transposed, d_right_transposed,
                       d_top, d_bottom, step, transposed_step,
                       size, oDeviceDst.data(), oDeviceDst.pitch(), pGraphcutState));
    fprintf(stderr, "nppiGraphcut_32f8u done\n");
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    float time;
    hipEventElapsedTime(&time, start, stop);
    fprintf(stderr, "Elapsed Time:  %f ms\n", time);

    // declare a host image object for an 8-bit grayscale image
    npp::ImageCPU_8u_C1 oHostAlpha(width, height);

    // convert graphcut result to 0/255 alpha image using new nppiCompareC_8u_C1R primitive (CUDA 5.0)
    npp::ImageNPP_8u_C1 oDeviceAlpha(width, height);
    NPP_CHECK_NPP(nppiCompareC_8u_C1R(oDeviceDst.data(), oDeviceDst.pitch(), 0, oDeviceAlpha.data(), oDeviceAlpha.pitch(), size,
                        NPP_CMP_GREATER));

    // and copy the result to host
    oDeviceAlpha.copyTo(oHostAlpha.data(), oHostAlpha.pitch());

    labels = new unsigned char[width*height];
    for (int i = 0; i < height; ++i)
        for (int j = 0; j < width; ++j)
            labels[i*width+j] = *oHostAlpha.data(i, j);

    checkCudaErrors(hipFree(d_terminals));
    checkCudaErrors(hipFree(d_top));
    checkCudaErrors(hipFree(d_bottom));
    checkCudaErrors(hipFree(d_left_transposed));
    checkCudaErrors(hipFree(d_right_transposed));
    checkCudaErrors(hipFree(pBuffer));
    nppiGraphcutFree(pGraphcutState);

    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
}

template<typename T>
T *transpose(T* mat, int rows, int cols)
{
    T* tmp = new T[rows*cols];
    for (int i = 0; i < rows; ++i)
        for (int j = 0; j < cols; ++j)
            tmp[i*cols+j] = mat[j*rows+i];
    return tmp;
}

template<typename T>
void transposeInPlace(T* mat, int rows, int cols)
{
    T* tmp = new T[rows*cols];
    for (int i = 0; i < rows; ++i)
        for (int j = 0; j < cols; ++j)
            tmp[i*cols+j] = mat[j*rows+i];
    memcpy(mat, tmp, sizeof(T)*rows*cols);
    delete [] tmp;
}

extern void mexFunction(int iNbOut, mxArray *pmxOut[],
    int iNbIn, const mxArray *pmxIn[])
{
    int width = (int)mxGetScalar(pmxIn[0]);
    int height = (int)mxGetScalar(pmxIn[1]);

    Npp32f *pTerminals = transpose((Npp32f*)mxGetData(pmxIn[2]), height, width);

    Npp32f* pLeftTransposed = transpose((Npp32f*)mxGetData(pmxIn[3]), width, height);
    for (int j = 0; j < height; ++j)
        if (pLeftTransposed[j] != 0)
            throw std::invalid_argument("pLeftTransposed[0][*] must be 0");

    Npp32f* pRightTransposed = transpose((Npp32f*)mxGetData(pmxIn[4]), width, height);
    for (int j = 0; j < height; ++j)
        if (pRightTransposed[(width-1)*height + j] != 0)
            throw std::invalid_argument("pRightTransposed[width-1][*] must be 0");  

    fprintf(stderr, "Assertions passed\n");  

    Npp32f* pTop = transpose((Npp32f*)mxGetData(pmxIn[5]), height, width);
    for (int j = 0 ; j < width; ++j)
        if (pTop[j] != 0)
            throw std::invalid_argument("pTop[0][*] must be 0");

    Npp32f* pBottom = transpose((Npp32f*)mxGetData(pmxIn[6]), height, width);
    for (int j = 0; j < width; ++j)
        if (pBottom[width*(height-1) + j] != 0)
            throw std::invalid_argument("pBottom[height-1][*] must be 0"); 

    pmxOut[0] = mxCreateNumericMatrix(height, width, mxINT8_CLASS, mxREAL);
    Npp8u *outmat = (Npp8u*)mxGetData(pmxOut[0]);

    graphCut(width, height, pTerminals, pLeftTransposed, pRightTransposed, pTop, pBottom, outmat);
    transposeInPlace(outmat, height, width); 

    delete [] pTerminals;
    delete [] pLeftTransposed;
    delete [] pRightTransposed;
    delete [] pTop;
    delete [] pBottom;
}
